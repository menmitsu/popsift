#include "hip/hip_runtime.h"
/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "sift_pyramid.h"
#include "sift_constants.h"
#include "gauss_filter.h"
#include "common/debug_macros.h"
#include "common/assist.h"
#include "common/clamp.h"

#include <iostream>
#include <stdio.h>

/* It makes no sense whatsoever to change this value */
#define PREV_LEVEL 3

namespace popsift {

namespace gauss {

namespace variableSpan {

namespace absoluteTexAddress {
__global__
void horiz( hipTextureObject_t src_data,
            Plane2D_float       dst_data,
            const int           dst_w,
            const int           dst_h,
            const int           dst_level )
{
    const int    src_level = dst_level - 1;
    const int    span      = d_gauss.inc.span[dst_level];
    const float* filter    = &popsift::d_gauss.inc.filter[dst_level*GAUSS_ALIGN];

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0.0f;

    #pragma unroll
    for( int offset = span; offset>0; offset-- ) {
        const float& g  = filter[offset];
        const float  v1 = readTex( src_data, off_x - offset, blockIdx.y, src_level );
        out += ( v1 * g );

        const float  v2 = readTex( src_data, off_x + offset, blockIdx.y, src_level );
        out += ( v2 * g );
    }
    const float& g  = filter[0];
    const float v3 = readTex( src_data, off_x, blockIdx.y, src_level );
    out += ( v3 * g );

    dst_data.ptr(blockIdx.y)[off_x] = out;
}

__device__ static inline
void vert_sub( hipTextureObject_t src_data,
               hipSurfaceObject_t dst_data,
               const int           dst_w,
               const int           dst_h,
               const int           dst_level,
               const int           span,
               const float*        filter )
{
    int block_x = blockIdx.x * blockDim.x;
    int block_y = blockIdx.y * blockDim.y;
    int idx     = threadIdx.x;
    int idy;

    float g;
    float val;
    float out = 0;

    for( int offset = span; offset>0; offset-- ) {
        g  = filter[offset];

        idy = threadIdx.y - offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );

        idy = threadIdx.y + offset;
        val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
        out += ( val * g );
    }

    g  = filter[0];
    idy = threadIdx.y;
    val = tex2D<float>( src_data, block_x + idx + 0.5f, block_y + idy + 0.5f );
    out += ( val * g );

    idx = block_x+threadIdx.x;
    idy = block_y+threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    surf2DLayeredwrite( out, dst_data, idx*4, idy, dst_level, hipBoundaryModeZero ); // dst_data.ptr(idy)[idx] = out;
}

__global__
void vert( hipTextureObject_t src_data,
           hipSurfaceObject_t dst_data,
           int                 dst_w,
           int                 dst_h,
           int                 dst_level )
{
    vert_sub( src_data, dst_data, dst_w, dst_h, dst_level, d_gauss.inc.span[dst_level], &popsift::d_gauss.inc.filter[dst_level*GAUSS_ALIGN] );
}

} // namespace absoluteTexAddress

namespace relativeTexAddress {

__device__
inline static void horiz_sub( hipTextureObject_t src_data,
                              Plane2D_float       dst_data,
                              float               shift,
                              int                 span,
                              float*              filter )
{
    const float dst_w  = dst_data.getWidth();
    const float dst_h  = dst_data.getHeight();
    const float read_y = ( blockIdx.y + shift ) / dst_h;

    const int off_x = blockIdx.x * blockDim.x + threadIdx.x;

    if( off_x >= dst_w ) return;

    float out = 0.0f;

    #pragma unroll
    for( int offset = span; offset>0; offset-- ) {
        const float& g  = filter[offset];
        const float read_x_l = ( off_x - offset );
        const float  v1 = tex2D<float>( src_data, ( read_x_l + shift ) / dst_w, read_y );
        out += ( v1 * g );

        const float read_x_r = ( off_x + offset );
        const float  v2 = tex2D<float>( src_data, ( read_x_r + shift ) / dst_w, read_y );
        out += ( v2 * g );
    }
    const float& g  = filter[0];
    const float read_x = off_x;
    const float v3 = tex2D<float>( src_data, ( read_x + shift ) / dst_w, read_y );
    out += ( v3 * g );

    dst_data.ptr(blockIdx.y)[off_x] = out * 255.0f;
}

__global__
void horiz( hipTextureObject_t src_data,
            Plane2D_float       dst_data,
            int                 octave,
            float               shift )
{
    // The first line creates level-0 octave-0 for the input image only.
    // Since we are computing the direct-downscaling gauss filter tables
    // and the first entry in that table is identical to the "normal"
    // table, we do not need a special case.
    // horiz( src_data, dst_data, shift, d_gauss.inc.span[0], &d_gauss.inc.filter[0*GAUSS_ALIGN] );
    horiz_sub( src_data,
               dst_data,
               shift,
               d_gauss.dd.span[octave],
               &d_gauss.dd.filter[octave*GAUSS_ALIGN] );
}

} // namespace relativeTexAddress

} // namespace variableSpan


__global__
void get_by_2_interpolate( hipTextureObject_t src_data,
                           const int           src_level,
                           hipSurfaceObject_t dst_data,
                           const int           dst_w,
                           const int           dst_h )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const float val = readTex( src_data, 2.0f * idx + 1.0f, 2.0f * idy + 1.0f, src_level );

    surf2DLayeredwrite( val, dst_data, idx*4, idy, 0, hipBoundaryModeZero ); // dst_data.ptr(idy)[idx] = val;
}

__global__
void get_by_2_pick_every_second( hipTextureObject_t src_data,
                                 const int           src_w,
                                 const int           src_h,
                                 const int           src_level,
                                 hipSurfaceObject_t dst_data,
                                 const int           dst_w,
                                 const int           dst_h )
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if( idx >= dst_w ) return;
    if( idy >= dst_h ) return;

    const int read_x = clamp( idx << 1, 0, src_w );
    const int read_y = clamp( idy << 1, 0, src_h );

    const float val = readTex( src_data, read_x, read_y, src_level );

    surf2DLayeredwrite( val, dst_data, idx*4, idy, 0, hipBoundaryModeZero ); // dst_data.ptr(idy)[idx] = val;
}


__global__
void make_dog( hipTextureObject_t src_data,
               hipSurfaceObject_t dog_data,
               const int           w,
               const int           h )
{
    const int idx   = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy   = blockIdx.y * blockDim.y + threadIdx.y;
    const int level = blockIdx.z;

    const float b = readTex( src_data, idx, idy, level+1 );
    const float a = readTex( src_data, idx, idy, level );
    const float c = b - a;

    surf2DLayeredwrite( c, dog_data, idx*4, idy, level, hipBoundaryModeZero );
}

} // namespace gauss

__host__
inline void Pyramid::horiz_from_input_image( const Config& conf, Image* base, int octave, hipStream_t stream, Config::SiftMode mode )
{
    Octave&   oct_obj = _octaves[octave];

    const int width   = oct_obj.getWidth();
    const int height  = oct_obj.getHeight();

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    float shift  = 0.5f;

    if( octave == 0 && ( mode == Config::PopSift || mode == Config::VLFeat ) ) {
        shift  = 0.5f * powf( 2.0f, conf.getUpscaleFactor() - octave );
    }

    gauss::variableSpan::relativeTexAddress::horiz
        <<<grid,block,0,stream>>>
        ( base->getInputTexture(),
          oct_obj.getIntermediateData( ),
          octave,
          shift );
}


__host__
inline void Pyramid::downscale_from_prev_octave( int octave, hipStream_t stream, Config::SiftMode mode )
{
    Octave&      oct_obj = _octaves[octave];
    Octave& prev_oct_obj = _octaves[octave-1];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 h_block( 64, 2 );
    dim3 h_grid;
    h_grid.x = (unsigned int)grid_divide( width,  h_block.x );
    h_grid.y = (unsigned int)grid_divide( height, h_block.y );

    switch( mode )
    {
    case Config::PopSift :
    case Config::VLFeat :
    case Config::OpenCV :
        gauss::get_by_2_pick_every_second
            <<<h_grid,h_block,0,stream>>>
            ( prev_oct_obj.getDataTexPoint( ),
              prev_oct_obj.getWidth(),
              prev_oct_obj.getHeight(),
              _levels-PREV_LEVEL,
              oct_obj.getDataSurface( ),
              oct_obj.getWidth(),
              oct_obj.getHeight() );
        break;
    default :
        gauss::get_by_2_interpolate
            <<<h_grid,h_block,0,stream>>>
            ( prev_oct_obj.getDataTexLinear( ),
              _levels-PREV_LEVEL,
              oct_obj.getDataSurface( ),
              oct_obj.getWidth(),
              oct_obj.getHeight() );
        break;
    }
}

__host__
inline void Pyramid::horiz_from_prev_level( int octave, int level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 128, 1 );
    dim3 grid;
    grid.x  = grid_divide( width,  128 );
    grid.y  = height;

    gauss::variableSpan::absoluteTexAddress::horiz
        <<<grid,block,0,stream>>>
        ( oct_obj.getDataTexPoint( ),
          oct_obj.getIntermediateData( ),
          oct_obj.getWidth(),
          oct_obj.getHeight(),
          level );
}

__host__
inline void Pyramid::vert_from_interm( int octave, int level, hipStream_t stream )
{
    Octave& oct_obj = _octaves[octave];

    /* waiting for any events is not necessary, it's in the same stream as horiz
     */

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 64, 2 );
    dim3 grid;
    grid.x = (unsigned int)grid_divide( width,  block.x );
    grid.y = (unsigned int)grid_divide( height, block.y );

    gauss::variableSpan::absoluteTexAddress::vert
        <<<grid,block,0,stream>>>
        ( oct_obj.getIntermDataTexPoint( ),
          oct_obj.getDataSurface( ),
          oct_obj.getWidth(),
          oct_obj.getHeight(),
          level );
}

__host__
inline void Pyramid::dogs_from_blurred( int octave, int max_level, hipStream_t stream )
{
    Octave&      oct_obj = _octaves[octave];

    const int width  = oct_obj.getWidth();
    const int height = oct_obj.getHeight();

    dim3 block( 128, 2 );
    dim3 grid;
    grid.x = grid_divide( width,  block.x );
    grid.y = grid_divide( height, block.y );
    grid.z = max_level - 1;

    gauss::make_dog
        <<<grid,block,0,stream>>>
        ( oct_obj.getDataTexPoint( ),
          oct_obj.getDogSurface( ),
          oct_obj.getWidth(),
          oct_obj.getHeight() );
}

/*************************************************************
 * V11: host side
 *************************************************************/
__host__
void Pyramid::build_pyramid( const Config& conf, Image* base )
{
#if (PYRAMID_PRINT_DEBUG==1)
    cerr << "Entering " << __FUNCTION__ << " with base image "  << endl
         << "    type size         : " << base->type_size << endl
         << "    aligned byte size : " << base->a_width << "x" << base->a_height << endl
         << "    pitch size        : " << base->pitch << "x" << base->a_height << endl
         << "    original byte size: " << base->u_width << "x" << base->u_height << endl
         << "    aligned pix size  : " << base->a_width/base->type_size << "x" << base->a_height << endl
         << "    original pix size : " << base->u_width/base->type_size << "x" << base->u_height << endl;
#endif // (PYRAMID_PRINT_DEBUG==1)

    hipDeviceSynchronize();

    for( uint32_t octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];
        hipStream_t stream  = oct_obj.getStream();

        if( ( conf.getScalingMode() == Config::ScaleDirect ) &&
            ( conf.getGaussMode() == Config::Fixed9 || conf.getGaussMode() == Config::Fixed15 ) ) {
            if( octave == 0 ) {
                make_octave( conf, base, oct_obj, stream, true );
            } else {
                horiz_from_input_image( conf, base, octave, stream, conf.getSiftMode() );
                vert_from_interm( octave, 0, stream );
                make_octave( conf, base, oct_obj, stream, false );
            }
        } else if( conf.getGaussMode() == Config::Fixed9 || conf.getGaussMode() == Config::Fixed15 ) {
            if( octave == 0 ) {
                make_octave( conf, base, oct_obj, stream, true );
            } else {
                Octave& prev_oct_obj = _octaves[octave-1];
                cuda::event_wait( prev_oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );

                downscale_from_prev_octave( octave, stream, conf.getSiftMode() );
                make_octave( conf, base, oct_obj, stream, false );
            }

            cuda::event_record( oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );
        } else if( conf.getScalingMode() == Config::ScaleDirect ) {
            for( int level=0; level<_levels; level++ ) {
                const int width  = oct_obj.getWidth();
                const int height = oct_obj.getHeight();

                if( level == 0 )
                {
                    horiz_from_input_image( conf, base, octave, stream, conf.getSiftMode() );
                    vert_from_interm( octave, level, stream );
                }
                else
                {
                    horiz_from_prev_level( octave, level, stream );
                    vert_from_interm( octave, level, stream );
                }
            }
        } else {
            for( int level=0; level<_levels; level++ ) {
                const int width  = oct_obj.getWidth();
                const int height = oct_obj.getHeight();

                if( level == 0 )
                {
                    if( octave == 0 )
                    {
                        horiz_from_input_image( conf, base, 0, stream, conf.getSiftMode() );
                        vert_from_interm( octave, 0, stream );
                    }
                    else
                    {
                        Octave& prev_oct_obj = _octaves[octave-1];
                        cuda::event_wait( prev_oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );

                        downscale_from_prev_octave( octave, stream, conf.getSiftMode() );
                    }
                }
                else
                {
                    horiz_from_prev_level( octave, level, stream );
                    vert_from_interm( octave, level, stream );

                    if( level == _levels - PREV_LEVEL ) {
                        cuda::event_record( oct_obj.getEventScaleDone(), stream, __FILE__, __LINE__ );
                    }
                }
            }

        }
    }
    for( int octave=_num_octaves-1; octave>=0; octave-- )
    {
        if( conf.getGaussMode() == Config::Fixed9 || conf.getGaussMode() == Config::Fixed15 ) {
        } else {
            Octave&      oct_obj = _octaves[octave];
            hipStream_t stream  = oct_obj.getStream();
            dogs_from_blurred( octave, _levels, stream );
        }
    }
    for( int octave=_num_octaves-1; octave>=0; octave-- )
    {
        Octave&      oct_obj = _octaves[octave];
        hipStream_t stream  = oct_obj.getStream();
        hipStreamSynchronize( stream );
    }
}

} // namespace popsift

