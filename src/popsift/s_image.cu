/*
 * Copyright 2016, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include "s_image.h"
#include <iostream>
#include <fstream>
#include "common/debug_macros.h"
#include "assist.h"
#include <stdio.h>
#include <assert.h>

using namespace std;

namespace popart {

Image::Image( size_t w, size_t h )
    : _w(w), _h(h)
{
    hipError_t err;

    _input_image_h.allocHost( w, h, popart::CudaAllocated );

    _input_image_d.allocDev( w, h );

    // cout << "Upscaled size of the input image: " << 2*w << "X" << 2*h << endl;

    /* initializing texture for upscaling
     */
    memset( &_input_image_texDesc, 0, sizeof(hipTextureDesc) );
    _input_image_texDesc.normalizedCoords = 1; // address 0..1 instead of 0..width/height
    _input_image_texDesc.addressMode[0]   = hipAddressModeClamp;
    _input_image_texDesc.addressMode[1]   = hipAddressModeClamp;
    _input_image_texDesc.addressMode[2]   = hipAddressModeClamp;
    _input_image_texDesc.readMode         = hipReadModeNormalizedFloat; // automatic conversion from uchar to float
    _input_image_texDesc.filterMode       = hipFilterModeLinear; // bilinear interpolation
    // _input_image_texDesc.filterMode       = hipFilterModePoint; // nearest neighbour mode

    memset( &_input_image_resDesc, 0, sizeof(hipResourceDesc) );
    _input_image_resDesc.resType                  = hipResourceTypePitch2D;
    _input_image_resDesc.res.pitch2D.devPtr       = _input_image_d.data;
    _input_image_resDesc.res.pitch2D.desc.f       = hipChannelFormatKindUnsigned;
    _input_image_resDesc.res.pitch2D.desc.x       = 8; // sizeof(uint8_t)*8
    _input_image_resDesc.res.pitch2D.desc.y       = 0;
    _input_image_resDesc.res.pitch2D.desc.z       = 0;
    _input_image_resDesc.res.pitch2D.desc.w       = 0;
    assert( _input_image_d.elemSize() == 1 );
    _input_image_resDesc.res.pitch2D.pitchInBytes = _input_image_d.step;
    _input_image_resDesc.res.pitch2D.width        = _input_image_d.getCols();
    _input_image_resDesc.res.pitch2D.height       = _input_image_d.getRows();

    err = hipCreateTextureObject( &_input_image_tex, &_input_image_resDesc, &_input_image_texDesc, 0 );
    POP_CUDA_FATAL_TEST( err, "Could not create texture object: " );
}

Image::~Image( )
{
    hipError_t err;
    err = hipDestroyTextureObject( _input_image_tex );
    POP_CUDA_FATAL_TEST( err, "Could not destroy texture object: " );

    _input_image_d.freeDev( );
    _input_image_h.freeHost( popart::CudaAllocated );
}

void Image::load( const Config& conf, const unsigned char* input )
{
    memcpy( _input_image_h.data, input, _w*_h );
    _input_image_h.memcpyToDevice( _input_image_d );
}

} // namespace popart

