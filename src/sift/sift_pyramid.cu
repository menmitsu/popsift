#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stdio.h>
#include <sys/stat.h>

#include "sift_pyramid.h"
#include "debug_macros.h"

#define PYRAMID_PRINT_DEBUG 0

using namespace std;

namespace popart {

/*************************************************************
 * CUDA device functions for printing debug information
 *************************************************************/

__global__
void py_print_corner_float( float* img, uint32_t pitch, uint32_t height, uint32_t level )
{
    const int xbase = 0;
    const int ybase = level * height + 0;
    for( int i=0; i<10; i++ ) {
        for( int j=0; j<10; j++ ) {
            printf("%3.3f ", img[(ybase+i)*pitch+xbase+j] );
        }
        printf("\n");
    }
    printf("\n");
}

__global__
void py_print_corner_float_transposed( float* img, uint32_t pitch, uint32_t height, uint32_t level )
{
    const int xbase = 0;
    const int ybase = level * height + 0;
    for( int i=0; i<10; i++ ) {
        for( int j=0; j<10; j++ ) {
            printf("%3.3f ", img[(ybase+j)*pitch+xbase+i] );
        }
        printf("\n");
    }
    printf("\n");
}

/*************************************************************
 * Callers for CUDA device functions that print debug information
 *************************************************************/

void Pyramid::debug_out_floats( float* data, uint32_t pitch, uint32_t height )
{
    py_print_corner_float
        <<<1,1>>>
        ( data,
          pitch,
          height,
          0 );

    test_last_error( __LINE__ );
}

void Pyramid::debug_out_floats_t( float* data, uint32_t pitch, uint32_t height )
{
    py_print_corner_float_transposed
        <<<1,1>>>
        ( data,
          pitch,
          height,
          0 );

    test_last_error( __LINE__ );
}

/*************************************************************
 * Host-sided debug function
 *************************************************************/

void Pyramid::test_last_error( int line )
{
    hipError_t err;
    hipDeviceSynchronize( );
    err = hipGetLastError();
    if( err != hipSuccess ) {
        printf("A problem in line %d, %s\n", line, hipGetErrorString(err) );
        exit( -__LINE__ );
    }
}

/*************************************************************
 * Debug output: write an octave/level to disk as PGM
 *************************************************************/

void Pyramid::download_and_save_array( const char* basename, uint32_t octave, uint32_t level )
{
    if( octave < _num_octaves ) {
        _octaves[octave].download_and_save_array( basename, octave, level );
    } else {
        cerr << "Octave " << octave << " does not exist" << endl;
        return;
    }
}

void Pyramid::download_descriptors( const Config& conf, uint32_t octave )
{
    _octaves[octave].downloadDescriptor( conf );
}

void Pyramid::save_descriptors( const Config& conf, const char* basename, uint32_t octave )
{
    struct stat st = {0};
    if (stat("dir-desc", &st) == -1) {
        mkdir("dir-desc", 0700);
    }
    ostringstream ostr;
    ostr << "dir-desc/desc-" << basename << "-o-" << octave << ".txt";
    ofstream of( ostr.str().c_str() );
    _octaves[octave].writeDescriptor( conf, of, true );

    if (stat("dir-fpt", &st) == -1) {
        mkdir("dir-fpt", 0700);
    }
    ostringstream ostr2;
    ostr2 << "dir-fpt/desc-" << basename << "-o-" << octave << ".txt";
    ofstream of2( ostr2.str().c_str() );
    _octaves[octave].writeDescriptor( conf, of2, false );
}

/*************************************************************
 * Pyramid constructor
 *************************************************************/

Pyramid::Pyramid( Config& config,
                  Image* base,
                  int width,
                  int height )
    : _num_octaves( config.octaves )
    , _levels( config.levels + 3 )
    , _scaling_mode( config.scaling_mode )
    , _gauss_group( config.gauss_group_size )
    , _assume_initial_blur( config.hasInitialBlur() )
    , _initial_blur( config.getInitialBlur() )
{
    // cerr << "Entering " << __FUNCTION__ << endl;

    _octaves = new Octave[_num_octaves];

    int w = width;
    int h = height;

    cout << "Size of the first octave's images: " << w << "X" << h << endl;

    for( int o=0; o<_num_octaves; o++ ) {
#if (PYRAMID_PRINT_DEBUG==1)
        printf("Allocating octave %u with width %u and height %u (%u levels)\n", o, w, h, _levels );
#endif // (PYRAMID_PRINT_DEBUG==1)
        _octaves[o].debugSetOctave( o );
        _octaves[o].alloc( w, h, _levels, _gauss_group );
        w = ceilf( w / 2.0f );
        h = ceilf( h / 2.0f );
    }
}

/*************************************************************
 * Pyramid destructor
 *************************************************************/

Pyramid::~Pyramid( )
{
    delete [] _octaves;
}

/*************************************************************
 * Build the pyramid in all levels, one octave
 *************************************************************/

void Pyramid::find_extrema( const Config& conf, Image* base )
{
    reset_extrema_mgmt( );

    build_pyramid( conf, base );

    find_extrema( conf );

    orientation( conf );

    descriptors_v1( conf );
}

void Pyramid::reset_extrema_mgmt( )
{
    for( int o=0; o<_num_octaves; o++ ) {
        _octaves[o].reset_extrema_mgmt( );
    }
}

} // namespace popart

