#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

#include "sift_pyramid.h"
#include "sift_constants.h"
#include "s_gradiant.h"
#include "assist.h"

#undef DESCRIPTORS_FROM_UNBLURRED_IMAGE

// override global setting
// #undef USE_DYNAMIC_PARALLELISM

/*************************************************************
 * V1: device side
 *************************************************************/

using namespace popart;
using namespace std;

__global__
void keypoint_descriptors( Extremum*     cand,
                           Descriptor*   descs,
                           Plane2D_float layer )
{
    const int width  = layer.getWidth();
    const int height = layer.getHeight();

    // int bidx = blockIdx.x & 0xf; // lower 4 bits of block ID
    const int ix   = threadIdx.y; // bidx & 0x3;       // lower 2 bits of block ID
    const int iy   = threadIdx.z; // bidx >> 2;        // next lowest 2 bits of block ID

    Extremum* ext = &cand[blockIdx.x];

    const float x    = ext->xpos;
    const float y    = ext->ypos;
    const float sig  = ext->sigma;
    const float ang  = ext->orientation;
    const float SBP  = fabsf(DESC_MAGNIFY * sig);

    if( SBP == 0 ) {
        return;
    }

    // const float cos_t = cosf(ang);
    // const float sin_t = sinf(ang);
    float cos_t;
    float sin_t;
    __sincosf( ang, &sin_t, &cos_t );

    const float csbp  = cos_t * SBP;
    const float ssbp  = sin_t * SBP;
    const float crsbp = cos_t / SBP;
    const float srsbp = sin_t / SBP;

    const float offsetptx = ix - 1.5f;
    const float offsetpty = iy - 1.5f;

    // The following 2 lines were the primary bottleneck of this kernel
    // const float ptx = csbp * offsetptx - ssbp * offsetpty + x;
    // const float pty = csbp * offsetpty + ssbp * offsetptx + y;
    const float ptx = fmaf( csbp, offsetptx, fmaf( -ssbp, offsetpty, x ) );
    const float pty = fmaf( csbp, offsetpty, fmaf(  ssbp, offsetptx, y ) );

    const float bsz = fabsf(csbp) + fabsf(ssbp);

    const int xmin = max(1,          (int)floorf(ptx - bsz));
    const int ymin = max(1,          (int)floorf(pty - bsz));
    const int xmax = min(width - 2,  (int)floorf(ptx + bsz));
    const int ymax = min(height - 2, (int)floorf(pty + bsz));

    const int wx = xmax - xmin + 1;
    const int hy = ymax - ymin + 1;
    const int loops = wx * hy;

    float dpt[9] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };
    // for (int i = 0; i < 9; i++) dpt[i] = 0.0f;

    for( int i = threadIdx.x; i < loops; i+=blockDim.x )
    {
        const int ii = i / wx + ymin;
        const int jj = i % wx + xmin;     

        const float dx = jj - ptx;
        const float dy = ii - pty;
        const float nx = fmaf( crsbp, dx,  srsbp * dy ); // crsbp * dx + srsbp * dy;
        const float ny = fmaf( crsbp, dy, -srsbp * dx ); // crsbp * dy - srsbp * dx;
        const float nxn = fabsf(nx);
        const float nyn = fabsf(ny);
        if (nxn < 1.0f && nyn < 1.0f) {
            const float2 mod_th = get_gradiant( jj, ii, layer );
            const float& mod    = mod_th.x;
            float        th     = mod_th.y;

            const float dnx = nx + offsetptx;
            const float dny = ny + offsetpty;
            const float ww  = __expf( -scalbnf(dnx*dnx + dny*dny, -3)); // speedup !
            // const float ww  = __expf(-0.125f * (dnx*dnx + dny*dny)); // speedup !
            const float wx  = 1.0f - nxn;
            const float wy  = 1.0f - nyn;
            const float wgt = ww * wx * wy * mod;

            th -= ang;
            th += ( th <  0.0f  ? M_PI2 : 0.0f ); //  if (th <  0.0f ) th += M_PI2;
            th -= ( th >= M_PI2 ? M_PI2 : 0.0f ); //  if (th >= M_PI2) th -= M_PI2;

            const float tth  = __fmul_ru( th, M_4RPI ); // th * M_4RPI;
            const int   fo0  = (int)floorf(tth);
            const float do0  = tth - fo0;             
            const float wgt1 = 1.0f - do0;
            const float wgt2 = do0;

            int fo  = fo0 % DESC_BINS;
            // if(fo < 8) {
                // maf: multiply-add
                // _ru - round to positive infinity equiv to froundf since always >=0
            dpt[fo]   = __fmaf_ru( wgt1, wgt, dpt[fo] );   // dpt[fo]   += (wgt1*wgt);
            dpt[fo+1] = __fmaf_ru( wgt2, wgt, dpt[fo+1] ); // dpt[fo+1] += (wgt2*wgt);
            // }
        }
        __syncthreads();
    }

    dpt[0] += dpt[8];

    /* reduction here */
    for (int i = 0; i < 8; i++) {
        dpt[i] += __shfl_down( dpt[i], 16 );
        dpt[i] += __shfl_down( dpt[i], 8 );
        dpt[i] += __shfl_down( dpt[i], 4 );
        dpt[i] += __shfl_down( dpt[i], 2 );
        dpt[i] += __shfl_down( dpt[i], 1 );
        dpt[i]  = __shfl     ( dpt[i], 0 );
    }

    // int hid    = blockIdx.x % 16;
    // int offset = hid*8;
    int offset = ( ( ( threadIdx.z << 2 ) + threadIdx.y ) << 3 ); // ( ( threadIdx.z * 4 ) + threadIdx.y ) * 8;

    Descriptor* desc = &descs[blockIdx.x];

    if( threadIdx.x < 8 ) {
        desc->features[offset+threadIdx.x] = dpt[threadIdx.x];
    }
}

__global__
void normalize_histogram( Descriptor* descs, int num_orientations )
{
    int offset = blockIdx.x * 32 + threadIdx.y;

    // all of these threads are useless
    if( blockIdx.x * 32 >= num_orientations ) return;

    bool ignoreme = ( offset >= num_orientations );

    offset = ( offset < num_orientations ) ? offset
                                           : num_orientations-1;
    Descriptor* desc = &descs[offset];

    float*  ptr1 = desc->features;
    float4* ptr4 = (float4*)ptr1;

    float4 descr;
    descr = ptr4[threadIdx.x];

#ifdef DESC_USE_ROOT_SIFT
    // root sift normalization
    float sum = descr.x + descr.y + descr.z + descr.w;

    sum += __shfl_down( sum, 16 );
    sum += __shfl_down( sum,  8 );
    sum += __shfl_down( sum,  4 );
    sum += __shfl_down( sum,  2 );
    sum += __shfl_down( sum,  1 );

    sum = __shfl( sum,  0 );

    /* multiplying with 512 is some scaling by convention */
    // sum = 512.0f / sum;
    // sum = __frcp_rn( scalbnf( sum, -9 ) );
    float val;
    val = 512.0f * __fsqrt_rn( __fdividef( descr.x, sum ) );
    descr.x = val;
    val = 512.0f * __fsqrt_rn( __fdividef( descr.y, sum ) );
    descr.y = val;
    val = 512.0f * __fsqrt_rn( __fdividef( descr.z, sum ) );
    descr.z = val;
    val = 512.0f * __fsqrt_rn( __fdividef( descr.w, sum ) );
    descr.w = val;

#else // not DESC_USE_ROOT_SIFT
    // OpenCV normalization

#undef HAVE_NORMF

#ifdef HAVE_NORMF
    float norm;

    if( threadIdx.x == 0 ) {
        norm = normf( 128, ptr1 );
    }

    norm = __shfl( norm,  0 );

    descr.x = min( descr.x, 0.2f*norm );
    descr.y = min( descr.y, 0.2f*norm );
    descr.z = min( descr.z, 0.2f*norm );
    descr.w = min( descr.w, 0.2f*norm );

    if( threadIdx.x == 0 ) {
        norm = 512.0f * rnormf( 128, ptr1 );
    }
#else
    float norm;

    norm = descr.x * descr.x
         + descr.y * descr.y
         + descr.z * descr.z
         + descr.w * descr.w;
    norm += __shfl_down( norm, 16 );
    norm += __shfl_down( norm,  8 );
    norm += __shfl_down( norm,  4 );
    norm += __shfl_down( norm,  2 );
    norm += __shfl_down( norm,  1 );
    if( threadIdx.x == 0 ) {
        norm = __fsqrt_rn( norm );
    }
    norm = __shfl( norm,  0 );

    descr.x = min( descr.x, 0.2f*norm );
    descr.y = min( descr.y, 0.2f*norm );
    descr.z = min( descr.z, 0.2f*norm );
    descr.w = min( descr.w, 0.2f*norm );

    norm = descr.x * descr.x
         + descr.y * descr.y
         + descr.z * descr.z
         + descr.w * descr.w;
    norm += __shfl_down( norm, 16 );
    norm += __shfl_down( norm,  8 );
    norm += __shfl_down( norm,  4 );
    norm += __shfl_down( norm,  2 );
    norm += __shfl_down( norm,  1 );
    if( threadIdx.x == 0 ) {
        norm = __fsqrt_rn( norm );
        norm = __fdividef( 512.0f, norm );
    }
#endif
    norm = __shfl( norm,  0 );

    descr.x = descr.x * norm;
    descr.y = descr.y * norm;
    descr.z = descr.z * norm;
    descr.w = descr.w * norm;

#endif // not DESC_USE_ROOT_SIFT

    if( not ignoreme ) {
        ptr4[threadIdx.x] = descr;
    }
}

__global__ void descriptor_starter( int*          extrema_counter,
                                    Extremum*     extrema,
                                    Descriptor*   descs,
                                    Plane2D_float layer )
{
#ifdef USE_DYNAMIC_PARALLELISM
    dim3 block;
    dim3 grid;
    grid.x  = *extrema_counter;

    if( grid.x == 0 ) return;

    // printf("Number of extrema after ori: %d\n", grid.x );

    block.x = 32;
    block.y = 4;
    block.z = 4;

    keypoint_descriptors
        <<<grid,block>>>
        ( extrema,
          descs,
          layer );

    // it may be good to start more threads, but this kernel
    // is too fast to be noticable in profiling

    grid.x  = grid_divide( *extrema_counter, 32 );
    block.x = 32;
    block.y = 32;
    block.z = 1;

    normalize_histogram
        <<<grid,block>>>
        ( descs, *extrema_counter );
#endif // not USE_DYNAMIC_PARALLELISM
}

/*************************************************************
 * V4: host side
 *************************************************************/
__host__
void Pyramid::descriptors_v1( )
{
#ifdef USE_DYNAMIC_PARALLELISM
    cerr << "Calling descriptors with dynamic parallelism" << endl;
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=1; level<_levels-2; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level+2);

#ifdef DESCRIPTORS_FROM_UNBLURRED_IMAGE
            Plane2D_float& data = oct_obj.getData( 0 );
#else // not DESCRIPTORS_FROM_UNBLURRED_IMAGE
            Plane2D_float& data = oct_obj.getData( level );
#endif // not DESCRIPTORS_FROM_UNBLURRED_IMAGE

            int* extrema_counters = oct_obj.getExtremaMgmtD();
            int* extrema_counter  = &extrema_counters[level];
            descriptor_starter
                <<<1,1,0,oct_str>>>
                ( extrema_counter,
                  oct_obj.getExtrema( level ),
                  oct_obj.getDescriptors( level ),
                  data );
        }
    }

    hipDeviceSynchronize();

    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave& oct_obj = _octaves[octave];
        oct_obj.readExtremaCount( );
    }
#else // not USE_DYNAMIC_PARALLELISM
    cerr << "Calling descriptors -no- dynamic parallelism" << endl;
    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=3; level<_levels; level++ ) {
            hipStreamSynchronize( oct_obj.getStream(level) );
        }

        // async copy of extrema from device to host
        oct_obj.readExtremaCount( );
    }

    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        int* num_orientations = oct_obj.getExtremaMgmtH();

        for( int level=1; level<_levels-2; level++ ) {
            dim3 block;
            dim3 grid;
            grid.x  = num_orientations[level];

            if( grid.x != 0 ) {
                block.x = 32;
                block.y = 4;
                block.z = 4;

#ifdef DESCRIPTORS_FROM_UNBLURRED_IMAGE
                Plane2D_float& data = oct_obj.getData( 0 );
#else // not DESCRIPTORS_FROM_UNBLURRED_IMAGE
                Plane2D_float& data = oct_obj.getData( level );
#endif // not DESCRIPTORS_FROM_UNBLURRED_IMAGE

                keypoint_descriptors
                    <<<grid,block,0,oct_obj.getStream(level+2)>>>
                    ( oct_obj.getExtrema( level ),
                      oct_obj.getDescriptors( level ),
                      data );

                grid.x  = grid_divide( num_orientations[level], 32 );
                block.x = 32;
                block.y = 32;
                block.z = 1;

                normalize_histogram
                    <<<grid,block,0,oct_obj.getStream(level+2)>>>
                    ( oct_obj.getDescriptors( level ),
                      num_orientations[level] );
            }
        }
    }

    hipDeviceSynchronize( );
#endif // not USE_DYNAMIC_PARALLELISM
}

