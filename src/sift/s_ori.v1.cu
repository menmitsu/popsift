#include "hip/hip_runtime.h"
#include "sift_pyramid.h"
#include "sift_constants.h"
#include "s_gradiant.h"
#include "debug_macros.h"

#include <math.h>
#include <stdio.h>
#include <inttypes.h>

using namespace popart;
using namespace std;

#undef V2_WITH_VLFEAT_SMOOTHING
#define V2_WITH_OPENCV_SMOOTHING

/*************************************************************
 * V1: device side
 *************************************************************/

__device__
inline float compute_angle( int bin, float hc, float hn, float hp )
{
    /* interpolate */
    float di = bin + 0.5f * (hn - hp) / (hc+hc-hn-hp);

    /* clamp */
    di = (di < 0) ? 
            (di + ORI_NBINS) : 
            ((di >= ORI_NBINS) ? (di - ORI_NBINS) : (di));

    float th = __fdividef( M_PI2 * di, ORI_NBINS ) - M_PI;
    // float th = ((M_PI2 * di) / ORI_NBINS);
    return th;
}

/*
 * Compute the keypoint orientations for each extremum
 * using 16 threads for each of them.
 */
__global__
void compute_keypoint_orientations_v1( Extremum*     extremum,
                                       int*          extrema_counter,
                                       Plane2D_float layer )
{
    uint32_t w   = layer.getWidth();
    uint32_t h   = layer.getHeight();

    // if( threadIdx.y >= mgmt->getCounter() ) return;

    Extremum* ext = &extremum[blockIdx.x];

    float hist[ORI_NBINS];
    for (int i = 0; i < ORI_NBINS; i++) hist[i] = 0.0f;

    /* keypoint fractional geometry */
    const float x    = ext->xpos;
    const float y    = ext->ypos;
    const float sig  = ext->sigma;

    /* orientation histogram radius */
    float  sigw = ORI_WINFACTOR * sig;
    int32_t rad  = (int)rintf((3.0f * sigw));

    float factor = __fdividef( -0.5f, (sigw * sigw) );
    int sq_thres = rad * rad;
    int32_t xmin = max(1,     (int32_t)floor(x - rad));
    int32_t xmax = min(w - 2, (int32_t)floor(x + rad));
    int32_t ymin = max(1,     (int32_t)floor(y - rad));
    int32_t ymax = min(h - 2, (int32_t)floor(y + rad));

    int wx = xmax - xmin + 1;
    int hy = ymax - ymin + 1;
    int loops = wx * hy;

    for(int i = threadIdx.x; i < loops; i+=ORI_V1_NUM_THREADS)
    {
        int yy = i / wx + ymin;
        int xx = i % wx + xmin;

        float grad;
        float theta;
        get_gradiant( grad,
                      theta,
                      xx,
                      yy,
                      layer );

        float dx = xx - x;
        float dy = yy - y;

        int sq_dist  = dx * dx + dy * dy;
        if (sq_dist <= sq_thres) {
            float weight = grad * expf(sq_dist * factor);

            int bidx = (int)rintf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );
            // int bidx = (int)roundf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );

            if( bidx > ORI_NBINS ) {
                printf("Crashing: bin %d theta %f :-)\n", bidx, theta);
            }

            bidx = (bidx == ORI_NBINS) ? 0 : bidx;

            hist[bidx] += weight;
        }
    }

    /* reduction here */
    for (int i = 0; i < ORI_NBINS; i++) {
        hist[i] += __shfl_down( hist[i], 8 );
        hist[i] += __shfl_down( hist[i], 4 );
        hist[i] += __shfl_down( hist[i], 2 );
        hist[i] += __shfl_down( hist[i], 1 );
        hist[i]  = __shfl( hist[i], 0 );
    }


    if(threadIdx.x != 0) return;

    for( int iter = 0; iter < 2; iter++ ) {
        float first = hist[0];
        float prev = hist[(ORI_NBINS - 1)];

        int bin;
        //0,35
        for( bin = 0; bin < ORI_NBINS - 1; bin++ ) {
            float temp = hist[bin];
            hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * hist[bin + 1];
            prev = temp;
        }

        hist[bin] = 0.25f * prev + 0.5f * hist[bin] + 0.25f * first;
        //z vprintf("val: %f, indx: %d\n", hist[bin], bin);
    }
	
    /* find histogram maximum */
    float maxh = NINF;
    int binh = 0;
    for (int bin = 0; bin < ORI_NBINS; bin++) {
        // maxh = fmaxf(maxh, hist[bin]);
        if (hist[bin] > maxh) {
            maxh = hist[bin];
            binh = bin;
        }
    }

    {
        float hc = hist[binh];
        float hn = hist[((binh + 1 + ORI_NBINS) % ORI_NBINS)];
        float hp = hist[((binh - 1 + ORI_NBINS) % ORI_NBINS)];
        float th = compute_angle(binh, hc, hn, hp);

        ext->orientation[0] = th;
    }

    /* find other peaks, boundary of 80% of max */
    int nangles = 1;

    for (int numloops = 1; numloops < ORI_NBINS; numloops++) {
        int bin = (binh + numloops) % ORI_NBINS;

        float hc = hist[bin];
        float hn = hist[((bin + 1 + ORI_NBINS) % ORI_NBINS)];
        float hp = hist[((bin - 1 + ORI_NBINS) % ORI_NBINS)];

        if (hc >= (0.8f * maxh) && hc > hn && hc > hp) {
            float th = compute_angle(bin, hc, hn, hp);

            ext->orientation[nangles] = th;

            nangles++;

            if( nangles == ORIENTATION_MAX_COUNT ) break;
        }
    }

    ext->num_ori = nangles;
}

/*
 * Compute the keypoint orientations for each extremum
 * using 16 threads for each of them.
 * direct curve fitting approach
 */
__global__
void compute_keypoint_orientations_v2( Extremum*     extremum,
                                       int*          extrema_counter,
                                       Plane2D_float layer,
                                       int*          d_number_of_blocks,
                                       int           number_of_blocks )
{
    uint32_t w   = layer.getWidth();
    uint32_t h   = layer.getHeight();

    // if( threadIdx.y >= mgmt->getCounter() ) return;

    Extremum* ext = &extremum[blockIdx.x];

    float hist[ORI_NBINS];
    for (int i = 0; i < ORI_NBINS; i++) hist[i] = 0.0f;

    /* keypoint fractional geometry */
    const float x    = ext->xpos;
    const float y    = ext->ypos;
    const float sig  = ext->sigma;

    /* orientation histogram radius */
    float  sigw = ORI_WINFACTOR * sig;
    int32_t rad  = (int)rintf((3.0f * sigw));

    float factor = __fdividef( -0.5f, (sigw * sigw) );
    int sq_thres  = rad * rad;

    int32_t xmin = max(1,     (int32_t)floor(x - rad));
    int32_t xmax = min(w - 2, (int32_t)floor(x + rad));
    int32_t ymin = max(1,     (int32_t)floor(y - rad));
    int32_t ymax = min(h - 2, (int32_t)floor(y + rad));

    int wx = xmax - xmin + 1;
    int hy = ymax - ymin + 1;
    int loops = wx * hy;

    for(int i = threadIdx.x; i < loops; i+=ORI_V1_NUM_THREADS)
    {
        int yy = i / wx + ymin;
        int xx = i % wx + xmin;

        float grad;
        float theta;
        get_gradiant( grad,
                      theta,
                      xx,
                      yy,
                      layer );

        float dx = xx - x;
        float dy = yy - y;

        int sq_dist  = dx * dx + dy * dy;
        if (sq_dist <= sq_thres) {
            float weight = grad * expf(sq_dist * factor);

            int bidx = (int)rintf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );
            // int bidx = (int)roundf( __fdividef( ORI_NBINS * (theta + M_PI), M_PI2 ) );

            if( bidx > ORI_NBINS ) {
                printf("Crashing: bin %d theta %f :-)\n", bidx, theta);
            }

            bidx = (bidx == ORI_NBINS) ? 0 : bidx;

            hist[bidx] += weight;
        }
    }

    /* reduction here */
    for (int i = 0; i < ORI_NBINS; i++) {
        hist[i] += __shfl_down( hist[i], 8 );
        hist[i] += __shfl_down( hist[i], 4 );
        hist[i] += __shfl_down( hist[i], 2 );
        hist[i] += __shfl_down( hist[i], 1 );
        hist[i]  = __shfl( hist[i], 0 );
    }


    if(threadIdx.x != 0) return;

    float xcoord[ORI_NBINS];
    float yval[ORI_NBINS];

    int   maxbin[ORIENTATION_MAX_COUNT];
    float y_max[ORIENTATION_MAX_COUNT];

    #pragma unroll
    for( int i=0; i<ORIENTATION_MAX_COUNT; i++ ) {
        maxbin[i] = -1;
        y_max[i] = -INFINITY;
    }

#ifdef V2_WITH_VLFEAT_SMOOTHING
    for( int i=0; i<3; i++ ) {
        for(int bin = 0; bin < ORI_NBINS; bin++) {
            int prev = bin == 0 ? ORI_NBINS-1 : bin-1;
            int next = bin == ORI_NBINS-1 ? 0 : bin+1;
            xcoord[bin] = ( hist[prev] + hist[bin] + hist[next] ) / 3.0f;
        }
        for(int bin = 0; bin < ORI_NBINS; bin++) {
            int prev = bin == 0 ? ORI_NBINS-1 : bin-1;
            int next = bin == ORI_NBINS-1 ? 0 : bin+1;
            hist[bin] = ( xcoord[prev] + xcoord[bin] + xcoord[next] ) / 3.0f;
        }
    }
#endif // V2_WITH_VLFEAT_SMOOTHING

#ifdef V2_WITH_OPENCV_SMOOTHING
    for(int bin = 0; bin < ORI_NBINS; bin++) {
        int prev2 = bin - 2;
        int prev1 = bin - 1;
        int next1 = bin + 1;
        int next2 = bin + 2;
        if( prev2 < 0 )          prev2 += ORI_NBINS;
        if( prev1 < 0 )          prev1 += ORI_NBINS;
        if( next1 >= ORI_NBINS ) next1 -= ORI_NBINS;
        if( next2 >= ORI_NBINS ) next2 -= ORI_NBINS;
        xcoord[bin] = (   hist[prev2] + hist[next2]
                      + ( hist[prev1] + hist[next1] ) * 4.0f
                      +   hist[bin] * 6.0f ) / 16.0f;
    }
    for(int bin = 0; bin < ORI_NBINS; bin++) {
        hist[bin] = xcoord[bin];
    }
#endif // V2_WITH_OPENCV_SMOOTHING

    for(int bin = 0; bin < ORI_NBINS; bin++) {
        // int prev = bin - 1;
        // if( prev < 0 ) prev = ORI_NBINS - 1;
        // int next = bin + 1;
        // if( next == ORI_NBINS ) next = 0;
        int prev = bin == 0 ? ORI_NBINS-1 : bin-1;
        int next = bin == ORI_NBINS-1 ? 0 : bin+1;

        if( hist[bin] > max( hist[prev], hist[next] ) ) {
            const float num = 3.0f * hist[prev] - 4.0f * hist[bin] + hist[next];
            const float denB = 2.0f * ( hist[prev] - 2.0f * hist[bin] + hist[next] );

            float newbin = __fdividef( num, denB ); // * M_PI/18.0f; // * 10.0f;
            if( newbin >= 0 && newbin <= 2 ) {
                xcoord[bin] = prev + newbin;
                yval[bin]   = -(num*num) / (4.0f * denB) + hist[prev];

#ifdef LOWE_ORIENTATION_MAX
                if( yval[bin] > y_max[0] ) {
                    y_max[2]  = y_max[1];
                    y_max[1]  = y_max[0];
                    y_max[0]  = yval[bin];
                    maxbin[2] = maxbin[1];
                    maxbin[1] = maxbin[0];
                    maxbin[0] = bin;
                } else if( yval[bin] > y_max[1] ) {
                    y_max[2]  = y_max[1];
                    y_max[1]  = yval[bin];
                    maxbin[2] = maxbin[1];
                    maxbin[1] = bin;
                } else if( yval[bin] > y_max[2] ) {
                    y_max[2]  = yval[bin];
                    maxbin[2] = bin;
                }
#else // LOWE_ORIENTATION_MAX
                if( yval[bin] > y_max[0] ) {
                    y_max[3]  = y_max[2];
                    y_max[2]  = y_max[1];
                    y_max[1]  = y_max[0];
                    y_max[0]  = yval[bin];
                    maxbin[3] = maxbin[2];
                    maxbin[2] = maxbin[1];
                    maxbin[1] = maxbin[0];
                    maxbin[0] = bin;
                } else if( yval[bin] > y_max[1] ) {
                    y_max[3]  = y_max[2];
                    y_max[2]  = y_max[1];
                    y_max[1]  = yval[bin];
                    maxbin[3] = maxbin[2];
                    maxbin[2] = maxbin[1];
                    maxbin[1] = bin;
                } else if( yval[bin] > y_max[2] ) {
                    y_max[3]  = y_max[2];
                    y_max[2]  = yval[bin];
                    maxbin[3] = maxbin[2];
                    maxbin[2] = bin;
                } else if( yval[bin] > y_max[3] ) {
                    y_max[3]  = yval[bin];
                    maxbin[3] = bin;
                }
#endif // LOWE_ORIENTATION_MAX
            }
        }
    }

    float chosen_bin = xcoord[maxbin[0]];
    if( chosen_bin >= ORI_NBINS ) chosen_bin -= ORI_NBINS;

    float th = __fdividef(M_PI2 * chosen_bin , ORI_NBINS) - M_PI;

    ext->orientation[0] = th;
    int angles = 1;

    for( int i=1; i<ORIENTATION_MAX_COUNT; i++ ) {
        if( y_max[i] < -1000.0f ) break; // this is a random number: no orientation can be this small

        if( y_max[i] < 0.8f * y_max[0] ) break;

        float chosen_bin = xcoord[maxbin[i]];
        if( chosen_bin >= ORI_NBINS ) chosen_bin -= ORI_NBINS;
        float th = __fdividef(M_PI2 * chosen_bin, ORI_NBINS) - M_PI;

        ext->orientation[i] = th;
        angles++;
    }

    ext->num_ori = angles;
}

/*************************************************************
 * V4: host side
 *************************************************************/
#ifdef USE_DYNAMIC_PARALLELISM // defined in_s_pyramid.h

__global__
void orientation_starter_v1( Extremum*     extremum,
                             int*          extrema_counter,
                             Plane2D_float layer )
{
    dim3 block;
    dim3 grid;
    grid.x  = *extrema_counter;
    block.x = ORI_V1_NUM_THREADS;

    if( grid.x != 0 ) {
        compute_keypoint_orientations_v1
            <<<grid,block>>>
            ( extremum,
              extrema_counter,
              layer );
    }
}

__global__
void orientation_starter_v2( Extremum*     extremum,
                             int*          extrema_counter,
                             Plane2D_float layer,
                             int*          d_number_of_blocks )
{
    dim3 block;
    dim3 grid;
    grid.x  = *extrema_counter;
    block.x = ORI_V1_NUM_THREADS;

    if( grid.x != 0 ) {
        compute_keypoint_orientations_v2
            <<<grid,block>>>
            ( extremum,
              extrema_counter,
              layer,
              d_number_of_blocks,
              grid.x * grid.y );
    }
}

__host__
void Pyramid::orientation_v1( )
{
    cerr << "Calling ori with dynamic parallelism" << endl;

    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        int*  orientation_num_blocks = oct_obj.getNumberOfOriBlocks( );

        for( int level=1; level<_levels-2; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level+2);

            int* extrema_counters = oct_obj.getExtremaMgmtD( );
            int* extrema_counter  = &extrema_counters[level];
            if( _bemap_orientation_mode ) {
                orientation_starter_v1
                    <<<1,1,0,oct_str>>>
                    ( oct_obj.getExtrema( level ),
                      extrema_counter,
                      oct_obj.getData( level ) );
            }  else {
                int*  num_blocks = &orientation_num_blocks[level];

                orientation_starter_v2
                    <<<1,1,0,oct_str>>>
                    ( oct_obj.getExtrema( level ),
                      extrema_counter,
                      oct_obj.getData( level ),
                      num_blocks );
            }
        }
    }
}

#else // not USE_DYNAMIC_PARALLELISM

__global__
void orientation_starter_v1( Extremum*,
                             int*,
                             Plane2D_float,
                             int* )
{
    /* dummy to make the linker happy */
}

__host__
void Pyramid::orientation_v1( )
{
    cerr << "Calling ori with -no- dynamic parallelism" << endl;

    for( int octave=0; octave<_num_octaves; octave++ ) {
        Octave&      oct_obj = _octaves[octave];

        for( int level=3; level<_levels; level++ ) {
            hipStreamSynchronize( oct_obj.getStream(level) );
        }

        oct_obj.readExtremaCount( );
        hipDeviceSynchronize( );

        int* h_num_extrema = oct_obj.getExtremaMgmtH();
        int* d_num_extrema = oct_obj.getExtremaMgmtD();
        int* orientation_num_blocks = oct_obj.getNumberOfOriBlocks( );

        for( int level=1; level<_levels-2; level++ ) {
            hipStream_t oct_str = oct_obj.getStream(level+2);

            dim3 block;
            dim3 grid;
            grid.x  = h_num_extrema[level];
            block.x = ORI_V1_NUM_THREADS;
            if( grid.x != 0 ) {
                if( _bemap_orientation_mode ) {
                    compute_keypoint_orientations_v1
                        <<<grid,block,0,oct_str>>>
                        ( oct_obj.getExtrema( level ),
                          &d_num_extrema[level],
                          oct_obj.getData( level ) );
                } else {
                    compute_keypoint_orientations_v2
                        <<<grid,block,0,oct_str>>>
                        ( oct_obj.getExtrema( level ),
                          &d_num_extrema[level],
                          oct_obj.getData( level ),
                          &orientation_num_blocks[level],
                          grid.x * grid.y );
                }
            }
        }
    }
}
#endif // not USE_DYNAMIC_PARALLELISM

