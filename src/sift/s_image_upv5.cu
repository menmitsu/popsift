
#include <hip/hip_runtime.h>
#if 0
#include "s_image.h"
#include "clamp.h"
#include "assist.h"

#include <iostream>
#include <sstream>
#include <map>

#undef FIND_BLOCK_SIZE

using namespace std;

namespace popart {

__global__
void p_upscale_opencv( Plane2D_float dst, cudaTextureObject_t src )
{
    int idx  = blockIdx.x * blockDim.x + threadIdx.x;
    int idy  = blockIdx.y * blockDim.y + threadIdx.y;
    if( idx >= dst.getCols() ) return;
    if( idy >= dst.getRows() ) return;
    const float src_x = (float(idx)+0.5f)/float(dst.getCols());
    const float src_y = (float(idy)+0.5f)/float(dst.getRows());
    float d = tex2D<float>( src, src_x, src_y );
    dst.ptr(idy)[idx] = d * 255.0f;
}

__global__
void p_upscale_vlfeat( Plane2D_float dst, cudaTextureObject_t src )
{
    int idx  = blockIdx.x * blockDim.x + threadIdx.x;
    int idy  = blockIdx.y * blockDim.y + threadIdx.y;
    if( idx >= dst.getCols() ) return;
    if( idy >= dst.getRows() ) return;
    const float src_x = (float(idx)+1.0f)/float(dst.getCols());
    const float src_y = (float(idy)+1.0f)/float(dst.getRows());
    float d = tex2D<float>( src, src_x, src_y );
    dst.ptr(idy)[idx] = d * 255.0f;
}

__global__
void p_upscale_popsift( Plane2D_float dst, cudaTextureObject_t src )
{
    int idx  = blockIdx.x * blockDim.x + threadIdx.x;
    int idy  = blockIdx.y * blockDim.y + threadIdx.y;
    if( idx >= dst.getCols() ) return;
    if( idy >= dst.getRows() ) return;
    const float src_x = (float(idx)+1.0f)/float(dst.getCols());
    const float src_y = (float(idy)+1.0f)/float(dst.getRows());
    float d = tex2D<float>( src, src_x, src_y );
    dst.ptr(idy)[idx] = d * 255.0f;
}

#ifdef FIND_BLOCK_SIZE
int condition[][2] = {
    // { 1, 1 }, { 8, 1 },
    { 32, 1 },
    { 64, 1 },
    { 128, 1 }, // this is the winner for GeForce GT 650M, CC 3.0 (MAC)
    { 256, 1 },
    { 1024, 1 },
    // { 1, 2 }, { 8, 2 },
    { 32, 2 }, { 64, 2 }, { 128, 2 }, { 256, 2 }, { 512, 2 },
    { 8, 8 }, { 32, 8 }, { 64, 8 }, { 128, 8 },
    { 32, 32 },
    { 0, 0 } };
__host__
void Image::upscale_v5( const Config& conf, cudaTextureObject_t & tex )
{
    std::map<float,string> logtimes;

    cudaEvent_t start;
    cudaEvent_t stop;
    cudaEventCreate( &start );
    cudaEventCreate( &stop );
    for( int cond=0; condition[cond][0]!=0; cond++ ) {
        int blockx = condition[cond][0];
        int blocky = condition[cond][1];

        int loops  = 100;

        cudaEventRecord( start, 0 );
        for( int i=0; i<loops; i++ ) {
            int gridx = grid_divide( this->_upscaled_image_d.getCols(), blockx );
            int gridy = grid_divide( this->_upscaled_image_d.getRows(), blocky );
            dim3 grid( gridx, gridy );
            dim3 block( blockx, blocky );

            if( conf.isVLFeatMode() ) {
                p_upscale_5_vlfeat<<<grid,block>>> ( this->_upscaled_image_d, tex );
            } else {
                p_upscale_5_opencv<<<grid,block>>> ( this->_upscaled_image_d, tex );
            }
        }
        cudaEventRecord( stop, 0 );
        cudaDeviceSynchronize( );
        float diff;
        cudaEventElapsedTime( &diff, start, stop );

        std::pair<float,string> datum;
        ostringstream ostr;
        ostr << "(" << blockx << ", " << blocky << ")";
        datum.first  = diff/loops;
        datum.second = ostr.str();
        logtimes.insert( datum );
    }

    std::map<float,string>::const_iterator it  = logtimes.begin();
    std::map<float,string>::const_iterator end = logtimes.end();
    for( ; it != end; it++ ) {
        const std::pair<float,string>& g = *it;
        cerr << "avg times: " << g.first
             << ": " << g.second
             << endl;
    }
    cudaEventDestroy( start );
    cudaEventDestroy( stop );
}
#else // not FIND_BLOCK_SIZE
__host__
void Image::upscale_v5( const Config& conf, cudaTextureObject_t & tex )
{
    dim3 block( 64, 2 );
    int gridx = grid_divide( this->_upscaled_image_d.getCols(), block.x );
    int gridy = grid_divide( this->_upscaled_image_d.getRows(), block.y );
    dim3 grid( gridx, gridy );

    switch( conf.getSiftMode() )
    {
    case Config::VLFeat :
        p_upscale_vlfeat
            <<<grid,block>>>
            ( this->_upscaled_image_d,
              tex );
        break;
    case Config::OpenCV :
        p_upscale_opencv
            <<<grid,block>>>
            ( this->_upscaled_image_d,
              tex );
        break;
    case Config::PopSift :
    default :
        p_upscale_popsift
            <<<grid,block>>>
            ( this->_upscaled_image_d,
              tex );
        break;
    }

    test_last_error( __FILE__,  __LINE__ );
}
#endif // not FIND_BLOCK_SIZE

} // namespace popart

#endif
